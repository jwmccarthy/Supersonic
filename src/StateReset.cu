#include "hip/hip_runtime.h"
#include "StateReset.cuh"

__device__ void resetBall(GameState* state, int simIdx) {
    // Reset ball to be motionless at center of field
    state->ballPosition[simIdx] = make_float4(0, 0, BALL_REST_Z, 0);
    state->ballVelocity[simIdx] = make_float4(0, 0, 0, 0);
    state->ballAngularVelocity[simIdx] = make_float4(0, 0, 0, 0);

    // Set ball rotation to identity
    state->ballRotationF[simIdx] = make_float4(1, 0, 0, 0);
    state->ballRotationR[simIdx] = make_float4(0, 1, 0, 0);
    state->ballRotationU[simIdx] = make_float4(0, 0, 1, 0);
}

__device__ void resetCar(GameState* state, int carIdx, const CarSpawn loc, bool invert) {
    // Get XY position, yaw angle
    float 
        x = invert ? -loc.x   : loc.x,
        y = invert ? -loc.y   : loc.y,
      yaw = invert ? -loc.yaw : loc.yaw;

    // Set car position
    state->carPosition[carIdx] = make_float4(x, y, CAR_REST_Z, 0);

    // Set car rotation
    auto yawRot = Mat3::FromEulerAngles(yaw, 0, 0);
    state->carRotationF[carIdx] = yawRot.f.v;
    state->carRotationR[carIdx] = yawRot.r.v;
    state->carRotationU[carIdx] = yawRot.u.v;

    // Reset car properties
    state->carVelocity[carIdx]        = CAR_RESET_DEFAULTS.velocity;
    state->carAngularVelocity[carIdx] = CAR_RESET_DEFAULTS.angularVelocity;
    state->carBoostAmount[carIdx]     = CAR_RESET_DEFAULTS.boostAmount;
    state->carDemolishTimer[carIdx]   = CAR_RESET_DEFAULTS.demolishTimer;
    state->carDemoCooldown[carIdx]    = CAR_RESET_DEFAULTS.demoCooldown;
    state->carIsOnGround[carIdx]      = CAR_RESET_DEFAULTS.isOnGround;
    state->carIsSupersonic[carIdx]    = CAR_RESET_DEFAULTS.isSupersonic;
    state->carHasJumped[carIdx]       = CAR_RESET_DEFAULTS.hasJumped;
    state->carHasDoubleJumped[carIdx] = CAR_RESET_DEFAULTS.hasDoubleJumped;
    state->carIsFlipping[carIdx]      = CAR_RESET_DEFAULTS.isFlipping;
    state->carIsBoosting[carIdx]      = CAR_RESET_DEFAULTS.isBoosting;
    state->carIsDemolished[carIdx]    = CAR_RESET_DEFAULTS.isDemolished;
}

__device__ void shuffleKickoffIndices(hiprandState_t &st, int* kickoffIndices, int teamSize) {
    // Clamp teamSize
    if (teamSize > NUM_KICKOFF_LOCATIONS) {
        teamSize = NUM_KICKOFF_LOCATIONS;
    }

    // Initialize indices
    for (int i = 0; i < NUM_KICKOFF_LOCATIONS; i++) {
        kickoffIndices[i] = i;
    }

    // Fisher-Yates initialize & shuffle
    for (int i = 0; i < teamSize; ++i) {
        int j = i + (hiprand(&st) % (NUM_KICKOFF_LOCATIONS - i));
        int temp = kickoffIndices[i];
        kickoffIndices[i] = kickoffIndices[j];
        kickoffIndices[j] = temp;
    }
}

__device__ void resetToKickoff(GameState* state, int simIdx) {
    const int nBlue   = state->numBlueCars;
    const int nOrange = state->numOrangeCars;
    const int nTotal  = state->carsPerSim;
    const int baseIdx = simIdx * nTotal;

    // Reset ball to center
    resetBall(state, simIdx);

    // Randomize kickoff indices
    hiprandState_t st = state->rngStates[simIdx];
    int maxTeamSize = max(nBlue, nOrange);
    int kickoffIndices[NUM_KICKOFF_LOCATIONS];
    shuffleKickoffIndices(st, kickoffIndices, maxTeamSize);

    // Reset blue & orange cars
    for (int i = 0; i < nTotal; i++) {
        int carIdx = baseIdx + i;
        bool orange = (i >= nBlue);

        // Reset index to mirror blue location
        int carSpawnIdx = orange ? (i - nBlue) : i;
        int kickoffIdx = kickoffIndices[carSpawnIdx];

        // Get location from randomized index
        const CarSpawn location = KICKOFF_LOCATIONS[kickoffIdx];
        
        // Reset car to associated random location
        resetCar(state, carIdx, location, orange);
    }

    state->rngStates[simIdx] = st;  // Write back for continued use
}