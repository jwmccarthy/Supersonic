#include "GameStateDevice.cuh"

GameStateDevice::GameStateDevice(int sims, int blues, int oranges, uint64_t seed) :
    m_simCount(sims),
    m_carsPerSim(blues + oranges),
    m_randomSeed(seed)
{
    // Allocate memory for each DeviceArray
    #define ALLOCATE_FIELD(type, name, count) \
        name.allocate(count);
    GAMESTATE_FIELDS(ALLOCATE_FIELD, m_simCount, m_carsPerSim)
    #undef ALLOCATE_FIELD

    // Create POD view of state
    h_view.simCount = m_simCount;
    h_view.numBlueCars = blues;
    h_view.numOrangeCars = oranges;
    h_view.carsPerSim = m_carsPerSim;
    h_view.randomSeed = m_randomSeed;

    // Set pointers to device array memory
    #define SETPTR_FIELD(type, name, count) \
        h_view.name = name.data();
    GAMESTATE_FIELDS(SETPTR_FIELD,,)
    #undef SETPTR_FIELD

    // Allocate memory for view on device
    CUDA_CHECK(hipMalloc(&d_view, sizeof(GameState)));

    // Copy the host staging struct to device
    CUDA_CHECK(hipMemcpy(d_view, &h_view, sizeof(GameState), hipMemcpyHostToDevice));
}

GameStateDevice::~GameStateDevice() {
    if (d_view) CUDA_CHECK(hipFree(d_view));
}