#include "InitializerKernels.cuh"
#include "RLEnvironment.cuh"

RLEnvironment::RLEnvironment(int sims, int blues, int oranges, uint64_t seed) :
    m_state(sims, blues, oranges, seed),
    m_stream(c10::cuda::getCurrentCUDAStream())
{
    // Set grid for initialization kernels
    int blockSize = 256;
    int gridSize = (sims + blockSize - 1) / blockSize;

    // Get POD game state struct
    GameState* d_state = m_state.view();

    // Initialize random seed
    printf("Initializing random states...\n");
    seedKernel<<<gridSize, blockSize, 0, m_stream>>>(d_state, seed);
    CUDA_CHECK(hipGetLastError());

    // Initialize ball and cars at kickoff locations
    printf("Initializing initial car & ball states...\n");
    resetToKickoffKernel<<<gridSize, blockSize, 0, m_stream>>>(d_state);
    CUDA_CHECK(hipGetLastError());

    // Activate boost pads
    printf("Activating boost pads...\n");
    m_state.boostPadIsActive.setValue(true, m_stream);

    // Calculate length of output buffer tensor
    int outputSize = m_state.getPhysicsStateLength();

    // Initialize output buffer tensor
    printf("Initializing tensor output buffer...\n");
    auto tensorOpts = torch::TensorOptions()
        .device(torch::kCUDA)
        .dtype(torch::kFloat32);
    m_output = torch::empty({outputSize}, tensorOpts);
    m_output.record_stream(m_stream);

    printf("Successfully initialized RLEnvironment\n");
}

torch::Tensor step(torch::Tensor actions) {
    return torch::empty({0});
}

torch::Tensor reset() {
    return torch::empty({0});
}