#include "hip/hip_runtime.h"
#include "Math.cuh"
#include "BroadPhaseGrid.cuh"

BroadPhaseGrid::BroadPhaseGrid(const std::string &meshPath,
                               int cellsX, int cellsY, int cellsZ,
                               float arenaX, float arenaY, float arenaZ)
:   m_numCellsX(cellsX),
    m_numCellsY(cellsY),
    m_numCellsZ(cellsZ)
{
    Mesh mesh;
    loadMeshObj(meshPath, mesh);
}

__device__ int4 BroadPhaseGrid::worldToCell(float4 point) const {
    int x = static_cast<int>((point.x - m_gridMinCorner.x) / m_gridExtents.x);
    int y = static_cast<int>((point.y - m_gridMinCorner.y) / m_gridExtents.y);
    int z = static_cast<int>((point.z - m_gridMinCorner.z) / m_gridExtents.z);

    x = clamp(x, 0, m_numCellsX - 1);
    y = clamp(y, 0, m_numCellsY - 1);
    z = clamp(z, 0, m_numCellsZ - 1);

    return {x, y, z};
}

template <typename Func>
__device__ void BroadPhaseGrid::forEachTriangle(float4 aabbMin, float4 aabbMax, Func&& func) const  {
    int4 startCell = worldToCell(aabbMin);
    int4 endCell   = worldToCell(aabbMax);

    for (int cellX = startCell.x; cellX <= endCell.x; ++cellX)
    for (int cellY = startCell.y; cellY <= endCell.y; ++cellY)
    for (int cellZ = startCell.z; cellZ <= endCell.z; ++cellZ)
    {
        int cellIdx = flattenIndex(cellX, cellY, cellZ);

        int triangleStart = m_cellOffsets[cellIdx];
        int triangleEnd   = m_cellOffsets[cellIdx + 1];

        for (int i = triangleStart; i < triangleEnd; ++i) {
            int triIdx = m_triangleIndices[i];

            int4 vertexIdx = m_triangles[triIdx];

            Triangle triangle {
                m_vertices[vertexIdx.x],
                m_vertices[vertexIdx.y],
                m_vertices[vertexIdx.z]
            };

            func(triangle);
        }
    }
}