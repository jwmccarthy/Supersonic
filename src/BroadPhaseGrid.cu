#include "hip/hip_runtime.h"
#include <numeric>
#include <algorithm>

#include "CudaMath.cuh"
#include "BroadPhaseGrid.cuh"

BroadPhaseGrid::BroadPhaseGrid(
    const std::string &meshPath,
    int cellsX, int cellsY, int cellsZ,
    float arenaX, float arenaY, float arenaZ
)
:   m_numCellsX(cellsX),
    m_numCellsY(cellsY),
    m_numCellsZ(cellsZ)
{
    // Min/max corner of arena extent
    m_gridMinCorner  = make_float4(-arenaX, -arenaY, 0, 0);
    m_gridMaxCorner  = make_float4( arenaX,  arenaY, arenaZ, 0);
    m_gridFullExtent = m_gridMaxCorner - m_gridMinCorner;

    // Pre-compute cell size (+ inverse)
    m_invCellSize = make_float4(
        m_numCellsX / m_gridFullExtent.x,
        m_numCellsY / m_gridFullExtent.y,
        m_numCellsZ / m_gridFullExtent.z, 0
    );

    // Read mesh data
    std::vector<float4> vertices;
    std::vector<int4>   triangles;
    loadMeshObj(meshPath, vertices, triangles);

    // Assign triangles to grid locations and construct offsets
    std::vector<int> cellOffsets;
    std::vector<int> triIndices;
    buildSpatialGrid(vertices, triangles, cellOffsets, triIndices);

    // Allocate space for pointers
    float4* d_vertices;
    int4*   d_triangles;
    int*    d_cellOffsets;
    int*    d_triIndices;

    size_t verticesSize = sizeof(float4) * vertices.size();
    size_t trianglesSize = sizeof(int4) * triangles.size();
    size_t cellOffsetsSize = sizeof(int) * cellOffsets.size();
    size_t triIndicesSize = sizeof(int) * triIndices.size();

    // Allocate device memory for triangle data
    CUDA_CHECK(hipMalloc(&d_vertices,    verticesSize));
    CUDA_CHECK(hipMalloc(&d_triangles,   trianglesSize));
    CUDA_CHECK(hipMalloc(&d_cellOffsets, cellOffsetsSize));
    CUDA_CHECK(hipMalloc(&d_triIndices,  triIndicesSize));

    // Move triangle information to device
    CUDA_CHECK(hipMemcpy(d_vertices,    vertices.data(),    verticesSize,    hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_triangles,   triangles.data(),   trianglesSize,   hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_cellOffsets, cellOffsets.data(), cellOffsetsSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_triIndices,  triIndices.data(),  triIndicesSize,  hipMemcpyHostToDevice));

    // Assign to constant pointers
    m_vertices = d_vertices;
    m_triangles = d_triangles;
    m_cellOffsets = d_cellOffsets;
    m_triIndices = d_triIndices;
}

void BroadPhaseGrid::buildSpatialGrid(
    const std::vector<float4>& vertices,
    const std::vector<int4>& triangles,
    std::vector<int>& cellOffsets,
    std::vector<int>& triIndices
) {
    int totalCells = m_numCellsX * m_numCellsY * m_numCellsZ;

    // Collect cell-triangle pairs
    std::vector<std::pair<int, int>> cellTriPairs;

    // Add triangles to cell lists
    for (int triIdx = 0; triIdx < triangles.size(); ++triIdx) {
        const int4& tri = triangles[triIdx];

        // Triangle vertices
        float4 v0 = vertices[tri.x];
        float4 v1 = vertices[tri.y];
        float4 v2 = vertices[tri.z];
        
        // Get triangle AABB
        float4 triMin = fminf(fminf(v0, v1), v2);
        float4 triMax = fmaxf(fmaxf(v0, v1), v2);

        // Convert points to cell indices
        int4 minCell = worldToCell(triMin);
        int4 maxCell = worldToCell(triMax);

        // Accumulate triangle indices for overlapped cells
        for (int x = minCell.x; x <= maxCell.x; x++)
        for (int y = minCell.y; y <= maxCell.y; y++)
        for (int z = minCell.z; z <= maxCell.z; z++)
        {
            int cellIdx = flattenIndex(x, y, z);
            cellTriPairs.push_back(std::make_pair(cellIdx, triIdx));
        }

        // Sort by cell index
        std::sort(cellTriPairs.begin(), cellTriPairs.end());

        // Build CSR format for triangle access by cell
        cellOffsets.assign(totalCells + 1, 0);
        triIndices.resize(cellTriPairs.size());

        for (size_t i = 0; i < cellTriPairs.size(); i++) {
            auto pair = cellTriPairs[i];
            cellOffsets[pair.first + 1]++;
            triIndices[i] = pair.second;
        }

        // Convert counts to offsets
        std::partial_sum(cellOffsets.begin(), cellOffsets.end(), cellOffsets.begin());
    }
}

__host__ __device__ int4 BroadPhaseGrid::worldToCell(float4 point) const {
    // Normalize to [0, numCells] range
    float4 normalized = (point - m_gridMinCorner) * m_invCellSize;
    
    // Convert to integer indices
    int x = static_cast<int>(floorf(normalized.x));
    int y = static_cast<int>(floorf(normalized.y));
    int z = static_cast<int>(floorf(normalized.z));
    
    // Clamp to valid range
    x = clamp(x, 0, m_numCellsX - 1);
    y = clamp(y, 0, m_numCellsY - 1);
    z = clamp(z, 0, m_numCellsZ - 1);
    
    return { x, y, z, 0 };
}

template <typename Func>
__device__ void BroadPhaseGrid::forEachTriangle(float4 aabbMin, float4 aabbMax, Func&& func) const  {
    int4 startCell = worldToCell(aabbMin);
    int4 endCell   = worldToCell(aabbMax);

    for (int cellX = startCell.x; cellX <= endCell.x; ++cellX)
    for (int cellY = startCell.y; cellY <= endCell.y; ++cellY)
    for (int cellZ = startCell.z; cellZ <= endCell.z; ++cellZ)
    {
        int cellIdx = flattenIndex(cellX, cellY, cellZ);

        int triangleStart = m_cellOffsets[cellIdx];
        int triangleEnd   = m_cellOffsets[cellIdx + 1];

        for (int i = triangleStart; i < triangleEnd; ++i) {
            int triIdx = m_triIndices[i];

            int4 vertexIdx = m_triangles[triIdx];

            Triangle triangle {
                m_vertices[vertexIdx.x],
                m_vertices[vertexIdx.y],
                m_vertices[vertexIdx.z]
            };

            func(triangle);
        }
    }
}